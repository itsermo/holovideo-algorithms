#include "hip/hip_runtime.h"
/*

*****************************JB: UNUSED. ADDED TO CHECK CUDA BUILDS VIA CMAKE. OK TO DELETE.

* Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample demonstrates Inter Process Communication
 *  features new to SDK 4.1 and uses one process per GPU for computation.
 * Note: Multiple processes per single device are possible but not recommended.
 *       In such cases, one should use IPC events for hardware synchronization.
 */

// Includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime includes
#include <hip/hip_runtime_api.h>

// CUDA utilities and system includes
#include <hip/hip_runtime_api.h>

int   *pArgc = NULL;
char **pArgv = NULL;

#define MAX_DEVICES          8
#define PROCESSES_PER_DEVICE 1
#define DATA_BUF_SIZE        4096

#ifdef __linux
#include <unistd.h>
#include <sched.h>
#include <sys/mman.h>
#include <sys/wait.h>
#include <linux/version.h>

typedef struct ipcCUDA_st
{
    int device;
    pid_t pid;
    hipIpcEventHandle_t eventHandle;
    hipIpcMemHandle_t memHandle;
} ipcCUDA_t;

typedef struct ipcDevices_st
{
    int count;
    int ordinals[MAX_DEVICES];
} ipcDevices_t;

typedef struct ipcBarrier_st
{
    int count;
    bool sense;
    bool allExit;
} ipcBarrier_t;

ipcBarrier_t *g_barrier = NULL;
bool          g_procSense;
int           g_processCount;

void procBarrier()
{
    int newCount = __sync_add_and_fetch(&g_barrier->count, 1);

    if (newCount == g_processCount)
    {
        g_barrier->count = 0;
        g_barrier->sense = !g_procSense;
    }
    else
    {
        while (g_barrier->sense == g_procSense)
        {
            if (!g_barrier->allExit)
            {
                sched_yield();
            }
            else
            {
                exit(EXIT_FAILURE);
            }
        }
    }

    g_procSense = !g_procSense;
}

// CUDA Kernel
__global__ void simpleKernel(int *dst, int *src, int num)
{
    // Dummy kernel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] / num;
}

void getDeviceCount(ipcDevices_t *devices)
{
    // We can't initialize CUDA before fork() so we need to spawn a new process

    pid_t pid = fork();

    if (0 == pid)
    {
        int i;
        int count, uvaCount = 0;
        int uvaOrdinals[MAX_DEVICES];
        printf("\nChecking for multiple GPUs...\n");
        checkCudaErrors(hipGetDeviceCount(&count));
        printf("CUDA-capable device count: %i\n", count);

        printf("\nSearching for UVA capable devices...\n");

        for (i = 0; i < count; i++)
        {
            hipDeviceProp_t prop;
            checkCudaErrors(hipGetDeviceProperties(&prop, i));

            if (prop.unifiedAddressing)
            {
                uvaOrdinals[uvaCount] = i;
                printf("> GPU%d = \"%15s\" IS capable of UVA\n", i, prop.name);
                uvaCount += 1;
            }

            if (prop.computeMode != hipComputeModeDefault)
            {
                printf("> GPU device must be in Compute Mode Default to run\n");
                printf("> Please use nvidia-smi to change the Compute Mode to Default\n");
                exit(EXIT_SUCCESS);
            }
        }

        devices->ordinals[0] = uvaOrdinals[0];

        if (uvaCount < 2)
        {
            devices->count = uvaCount;
            exit(EXIT_SUCCESS);
        }

        // Check possibility for peer accesses, relevant to our tests
        printf("\nChecking GPU(s) for support of peer to peer memory access...\n");
        devices->count = 1;
        int canAccessPeer_0i, canAccessPeer_i0;

        for (i = 1; i < uvaCount; i++)
        {
            checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeer_0i, uvaOrdinals[0], uvaOrdinals[i]));
            checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeer_i0, uvaOrdinals[i], uvaOrdinals[0]));

            if (canAccessPeer_0i*canAccessPeer_i0)
            {
                devices->ordinals[devices->count] = uvaOrdinals[i];
                printf("> Two-way peer access between GPU%d and GPU%d: YES\n", devices->ordinals[0], devices->ordinals[devices->count]);
                devices->count += 1;
            }
        }

        exit(EXIT_SUCCESS);
    }
    else
    {
        int status;
        waitpid(pid, &status, 0);
        assert(!status);
    }
}

inline bool IsAppBuiltAs64()
{
#if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
    return 1;
#else
    return 0;
#endif
}

void runTestMultiKernel(ipcCUDA_t *s_mem, int index)
{
    /*
     * a) Process 0 loads a reference buffer into GPU0 memory
     * b) Other processes launch a kernel on the GPU0 memory using P2P
     * c) Process 0 checks the resulting buffer
     */

    // memory buffer in gpu
    int *d_ptr;

    // reference buffer in host memory  (do in all processes for rand() consistency)
    int h_refData[DATA_BUF_SIZE];

    for (int i = 0; i < DATA_BUF_SIZE; i++)
    {
        h_refData[i] = rand();
    }

    checkCudaErrors(hipSetDevice(s_mem[index].device));

    if (index == 0)
    {
        printf("\nLaunching kernels...\n");
        // host memory buffer for checking results
        int h_results[DATA_BUF_SIZE * MAX_DEVICES * PROCESSES_PER_DEVICE];

        hipEvent_t event[MAX_DEVICES * PROCESSES_PER_DEVICE];
        checkCudaErrors(hipMalloc((void **) &d_ptr, DATA_BUF_SIZE * g_processCount * sizeof(int)));
        checkCudaErrors(hipIpcGetMemHandle((hipIpcMemHandle_t *) &s_mem[0].memHandle, (void *) d_ptr));
        checkCudaErrors(hipMemcpy((void *) d_ptr, (void *) h_refData, DATA_BUF_SIZE * sizeof(int), hipMemcpyHostToDevice));

        // b.1: wait until all event handles are created in other processes
        procBarrier();

        for (int i = 1; i < g_processCount; i++)
        {
            checkCudaErrors(hipIpcOpenEventHandle(&event[i], s_mem[i].eventHandle));
        }

        // b.2: wait until all kernels launched and events recorded
        procBarrier();

        for (int i = 1; i < g_processCount; i++)
        {
            checkCudaErrors(hipEventSynchronize(event[i]));
        }

        // b.3
        procBarrier();

        checkCudaErrors(hipMemcpy(h_results, d_ptr + DATA_BUF_SIZE,
                                   DATA_BUF_SIZE * (g_processCount - 1) * sizeof(int), hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(d_ptr));
        printf("Checking test results...\n");

        for (int n = 1; n < g_processCount; n++)
        {
            for (int i = 0; i < DATA_BUF_SIZE; i++)
            {
                if (h_refData[i]/(n + 1) != h_results[(n-1) * DATA_BUF_SIZE + i])
                {
                    fprintf(stderr, "Data check error at index %d in process %d!: %i,    %i\n",i,
                            n, h_refData[i], h_results[(n-1) * DATA_BUF_SIZE + i]);
                    g_barrier->allExit = true;
                    exit(EXIT_FAILURE);
                }
            }
        }
    }
    else
    {
        hipEvent_t event;
        checkCudaErrors(hipEventCreate(&event, hipEventDisableTiming | hipEventInterprocess));
        checkCudaErrors(hipIpcGetEventHandle((hipIpcEventHandle_t *) &s_mem[index].eventHandle, event));

        // b.1: wait until proc 0 initializes device memory
        procBarrier();

        checkCudaErrors(hipIpcOpenMemHandle((void **) &d_ptr, s_mem[0].memHandle,
                                             hipIpcMemLazyEnablePeerAccess));
        printf("> Process %3d: Run kernel on GPU%d, taking source data from and writing results to process %d, GPU%d...\n",
               index, s_mem[index].device, 0, s_mem[0].device);
        const dim3 threads(512, 1);
        const dim3 blocks(DATA_BUF_SIZE / threads.x, 1);
        simpleKernel<<<blocks, threads>>> (d_ptr + index *DATA_BUF_SIZE, d_ptr, index + 1);
        checkCudaErrors(hipEventRecord(event));

        // b.2
        procBarrier();

        checkCudaErrors(hipIpcCloseMemHandle(d_ptr));

        // b.3: wait till all the events are used up by proc g_processCount - 1
        procBarrier();

        checkCudaErrors(hipEventDestroy(event));
    }
}
#endif


void prepareCUDA() {
	int count = 0;
    printf("\nChecking for multiple GPUs...\n");
    checkCudaErrors(hipGetDeviceCount(&count));
    printf("CUDA-capable device count: %i\n", count);
}

