#include "hip/hip_runtime.h"
#include "dscp4-fringe-cuda.h"

#include <hip/hip_runtime.h>

#ifdef WIN32
#include <Windows.h>
#endif

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

const int N = 16;
const int blocksize = 16;

#include <stdio.h>

__global__ void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

__global__ void computeFringe(void * fringeDataOut, void * rgbaIn, void * depthIn)
{
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	((int*)fringeDataOut)[i] = 0;
	((int*)fringeDataOut)[j] = 0;
}

char * dscp4_fringe_cuda_HelloWorld()
{
	char *a = (char*)malloc(N);
	a[0] = 'H';
	a[1] = 'e';
	a[2] = 'l';
	a[3] = 'l';
	a[4] = 'o';
	a[5] = ' ';
	a[6] = '\0';
	a[7] = '\0';
	a[8] = '\0';
	a[9] = '\0';
	a[10] = '\0';
	a[11] = '\0';
	//a[12] = '\0';
	//a[13] = '\0';
	//a[14] = '\0';
	//a[15] = '\0';

	//strcpy	"Hello \0\0\0\0\0\0";
	int b[N] = { 15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	//printf("%s", a);

	hipMalloc((void**)&ad, csize);
	hipMalloc((void**)&bd, isize);
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	hello << <dimGrid, dimBlock >> >(ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(bd);

	return a;
	//printf("%s\n", a);
	//return EXIT_SUCCESS;
};

dscp4_fringe_cuda_context_t* dscp4_fringe_cuda_CreateContext(dscp4_fringe_context_t* fringeContext)
{
	hipError_t error = hipSuccess;
	dscp4_fringe_cuda_context_t* cudaContext = (dscp4_fringe_cuda_context_t*)malloc(sizeof(dscp4_fringe_cuda_context_t));
	cudaContext->fringe_context = fringeContext;

	error = hipGetDeviceCount(&cudaContext->num_gpus);
	
	if (error != hipSuccess)
	{
		free(cudaContext);
		return NULL;
	}

	cudaContext->gpu_properties = (struct hipDeviceProp_t*)malloc(sizeof(struct hipDeviceProp_t)*cudaContext->num_gpus);

	for (int i = 0; i < cudaContext->num_gpus; i++)
	{
		error = cudaGLSetGLDevice(i);
		error = hipGetDeviceProperties(&cudaContext->gpu_properties[i], i);
	}

	error = hipGraphicsGLRegisterBuffer(&cudaContext->stereogram_rgba_cuda_resource, cudaContext->fringe_context->stereogram_gl_rgba_buf_in, hipGraphicsRegisterFlagsReadOnly);
	error = hipGraphicsGLRegisterBuffer(&cudaContext->stereogram_depth_cuda_resource, cudaContext->fringe_context->stereogram_gl_depth_buf_in, hipGraphicsRegisterFlagsReadOnly);

	//error = hipGraphicsGLRegisterImage(&cudaContext->fringe_cuda_resources, cudaContext->fringe_context->fringe_gl_buf_out[0], GL_TEXTURE_2D, cudaGraphicsMapFlagsWriteDiscard);

	cudaContext->fringe_cuda_resources = (struct hipGraphicsResource**)malloc(sizeof(void*)*cudaContext->fringe_context->display_options.num_heads / 2);

	for (unsigned int i = 0; i < cudaContext->fringe_context->display_options.num_heads / 2; i++)
	{
		error = hipGraphicsGLRegisterBuffer(&cudaContext->fringe_cuda_resources[i], cudaContext->fringe_context->fringe_gl_buf_out[i], hipGraphicsRegisterFlagsWriteDiscard);
	}

	return cudaContext;
};

void dscp4_fringe_cuda_DestroyContext(dscp4_fringe_cuda_context_t** cudaContext)
{

	for (unsigned int i = 0; i < (*cudaContext)->fringe_context->display_options.num_heads / 2; i++)
	{
		hipGraphicsUnregisterResource((*cudaContext)->fringe_cuda_resources[i]);
	}

	hipGraphicsUnregisterResource((*cudaContext)->stereogram_depth_cuda_resource);
	hipGraphicsUnregisterResource((*cudaContext)->stereogram_rgba_cuda_resource);

	if (*cudaContext != NULL)
	{
		if ((*cudaContext)->fringe_cuda_resources)
		{
			free((*cudaContext)->fringe_cuda_resources);
			(*cudaContext)->fringe_cuda_resources = NULL;
		}

		if ((*cudaContext)->gpu_properties)
		{
			free((*cudaContext)->gpu_properties);
			(*cudaContext)->gpu_properties = NULL;
		}

		free(*cudaContext);
		*cudaContext = NULL;
	}
};

void dscp4_fringe_cuda_ComputeFringe(dscp4_fringe_cuda_context_t* cudaContext)
{
	// The total number of wafels in one frame
	const int NUM_WAFELS = cudaContext->fringe_context->algorithm_options.num_wafels_per_scanline *
		cudaContext->fringe_context->algorithm_options.num_scanlines;
	
	// The size (in bytes) per wafel
	const size_t WAFEL_SIZE = cudaContext->fringe_context->display_options.head_res_x *
		cudaContext->fringe_context->display_options.head_res_y *
		cudaContext->fringe_context->display_options.num_heads * sizeof(char) * 3 /
		NUM_WAFELS;

	void **output;
	size_t * outputSizes;

	void * rgbaPtr;
	void * depthPtr;
	size_t rgbaSize;
	size_t depthSize;

	hipError_t error = hipSuccess;

	error = hipGraphicsMapResources(1, &cudaContext->stereogram_rgba_cuda_resource, 0);
	error = hipGraphicsMapResources(1, &cudaContext->stereogram_depth_cuda_resource, 0);

	hipGraphicsResourceGetMappedPointer((void**)&rgbaPtr, &rgbaSize, cudaContext->stereogram_rgba_cuda_resource);
	hipGraphicsResourceGetMappedPointer((void**)&depthPtr, &depthSize, cudaContext->stereogram_depth_cuda_resource);

	output = (void**)malloc(sizeof(void*)* cudaContext->fringe_context->display_options.num_heads / 2);
	outputSizes = (size_t*)malloc(sizeof(size_t) * cudaContext->fringe_context->display_options.num_heads / 2);

	for (int i = 0; i < cudaContext->fringe_context->display_options.num_heads / 2; i++)
	{
		error = hipGraphicsMapResources(1, (hipGraphicsResource_t*)(&cudaContext->fringe_cuda_resources[i]), 0);
		error = hipGraphicsResourceGetMappedPointer(&output[i], &outputSizes[i], cudaContext->fringe_cuda_resources[i]);
	}

	//error = hipMemset(output[0], 255, outputSizes[0]);

	// run kernel here
	dim3 threadsPerBlock(
		cudaContext->fringe_context->algorithm_options.num_wafels_per_scanline,
		cudaContext->fringe_context->algorithm_options.num_scanlines);
	dim3 numBlocks(cudaContext->fringe_context->algorithm_options.num_wafels_per_scanline * 4 / threadsPerBlock.x,
		cudaContext->fringe_context->algorithm_options.num_scanlines * 4 / threadsPerBlock.y);
	computeFringe <<<numBlocks, threadsPerBlock >>>(output[0], rgbaPtr, depthPtr);


	//write texture outputs here

	for (int i = 0; i < cudaContext->fringe_context->display_options.num_heads / 2; i++)
	{
		error = hipGraphicsUnmapResources(1, (hipGraphicsResource_t*)(&cudaContext->fringe_cuda_resources[i]), 0);
	}

	error = hipGraphicsUnmapResources(1, &cudaContext->stereogram_rgba_cuda_resource, 0);
	error = hipGraphicsUnmapResources(1, &cudaContext->stereogram_depth_cuda_resource, 0);

	free(output);
	free(outputSizes);
};