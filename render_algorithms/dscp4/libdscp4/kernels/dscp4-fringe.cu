#include "hip/hip_runtime.h"
#include "dscp4-fringe-cuda.h"
#include <hip/hip_runtime.h>

const int N = 16;
const int blocksize = 16;

#include <stdio.h>

__global__ void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}


void dscp4_fringe_cuda_HelloWorld()
{
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = { 15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc((void**)&ad, csize);
	hipMalloc((void**)&bd, isize);
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	hello << <dimGrid, dimBlock >> >(ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(bd);

	printf("%s\n", a);
	//return EXIT_SUCCESS;
};

dscp4_fringe_cuda_context_t* dscp4_fringe_cuda_CreateContext(dscp4_fringe_context_t fringeContext)
{
	dscp4_fringe_cuda_context_t* context = (dscp4_fringe_cuda_context_t*)malloc(sizeof(dscp4_fringe_cuda_context_t));
	context->fringe_context = fringeContext;

	return context;
};

void dscp4_fringe_cuda_DestroyContext(dscp4_fringe_cuda_context_t** cudaContext)
{
	if (*cudaContext != NULL)
	{
		free(*cudaContext);
		*cudaContext = NULL;
	}
};

void dscp4_fringe_cuda_ComputeFringe()
{

};