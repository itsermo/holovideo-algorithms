#include "hip/hip_runtime.h"
#include "dscp4-fringe-cuda.h"

#include <hip/hip_runtime.h>

#ifdef WIN32
#include <Windows.h>
#endif

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>

texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> viewset_color_in;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> * framebuffer_tex_out;

__global__ void computeFringe(
	unsigned char* framebuffer_out,
	const float* viewset_depth_in,
	const unsigned int num_wafels_per_scanline,
	const unsigned int num_scanlines,
	const unsigned int viewset_res_x,
	const unsigned int viewset_res_y,
	const unsigned int viewset_num_tiles_x,
	const unsigned int viewset_num_tiles_y,
	const unsigned int framebuffer_res_x,
	const unsigned int framebuffer_res_y,
	unsigned char* wafel_buffer,
	float* wafel_position,
	const float K_R,
	const float K_G,
	const float K_B,
	const float UPCONVERT_CONST_R,
	const float UPCONVERT_CONST_G,
	const float UPCONVERT_CONST_B,
	const unsigned int NUM_SAMPLES_PER_WAFEL,
	const float SAMPLE_PITCH,
	const float Z_SPAN,
	const float Z_OFFSET,
	const unsigned int NUM_AOM_CHANNELS,
	const unsigned int HEAD_RES_Y_SPEC,
	const unsigned int NUM_BUFFERS
	);

dscp4_fringe_cuda_context_t* dscp4_fringe_cuda_CreateContext(dscp4_fringe_context_t* fringeContext)
{
	hipError_t error = hipSuccess;
	dscp4_fringe_cuda_context_t* cudaContext = (dscp4_fringe_cuda_context_t*)malloc(sizeof(dscp4_fringe_cuda_context_t));
	cudaContext->fringe_context = fringeContext;

	error = hipGetDeviceCount(&cudaContext->num_gpus);

	
	if (error != hipSuccess)
	{
		printf("ERROR Could not get CUDA device count -- Are there any CUDA devices present?");
		free(cudaContext);
		return NULL;
	}

	cudaContext->gpu_properties = (struct hipDeviceProp_t*)malloc(sizeof(struct hipDeviceProp_t)*cudaContext->num_gpus);

	for (int i = 0; i < cudaContext->num_gpus; i++)
	{
		error = cudaGLSetGLDevice(i);
		error = hipGetDeviceProperties(&cudaContext->gpu_properties[i], i);
	}

	error = hipGraphicsGLRegisterImage(&cudaContext->stereogram_rgba_cuda_resource, cudaContext->fringe_context->stereogram_gl_fbo_color, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
	if (error != hipSuccess)
		printf("ERROR Could not register viewset OpenGL RGBA texture");

	error = hipGraphicsGLRegisterBuffer(&cudaContext->stereogram_depth_cuda_resource, cudaContext->fringe_context->stereogram_gl_depth_buf_in, hipGraphicsRegisterFlagsReadOnly);
	if (error != hipSuccess)
		printf("ERROR Could not register viewset OpenGL DEPTH texture with CUDA");

	cudaContext->fringe_cuda_resources = (struct hipGraphicsResource**)malloc(sizeof(void*)*cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers);

	error = hipMalloc((void**)framebuffer_tex_out, sizeof(framebuffer_tex_out) * cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers);
	if (error)
		printf("ERROR Could not alloc CUDA framebuffer textures");

	for (unsigned int i = 0; i < cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers; i++)
	{
		//error = hipGraphicsGLRegisterBuffer(&cudaContext->fringe_cuda_resources[i], cudaContext->fringe_context->fringe_gl_buf_out[i], hipGraphicsRegisterFlagsWriteDiscard);
		error = hipGraphicsGLRegisterImage(&cudaContext->fringe_cuda_resources[i], cudaContext->fringe_context->fringe_gl_tex_out[i], GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard);
		if (error)
			printf("ERROR Could not register CUDA image framebuffer texture objects");
	}

	error = hipMalloc(&cudaContext->spec_buffer, cudaContext->fringe_context->display_options.head_res_x_spec * cudaContext->fringe_context->display_options.head_res_y_spec * cudaContext->fringe_context->display_options.num_heads * 4);
	if (error != hipSuccess)
		printf("ERROR Could not alloc CUDA megabuffer");

	error = hipMalloc(&cudaContext->wafel_buffers, cudaContext->fringe_context->algorithm_options.cache.num_samples_per_wafel * cudaContext->fringe_context->algorithm_options.num_wafels_per_scanline * cudaContext->fringe_context->algorithm_options.num_scanlines * sizeof(unsigned char));
	error = hipMalloc(&cudaContext->wafel_positions, cudaContext->fringe_context->algorithm_options.cache.num_samples_per_wafel * cudaContext->fringe_context->algorithm_options.num_wafels_per_scanline * cudaContext->fringe_context->algorithm_options.num_scanlines * sizeof(float));

	return cudaContext;
};

void dscp4_fringe_cuda_DestroyContext(dscp4_fringe_cuda_context_t** cudaContext)
{

	hipFree((*cudaContext)->wafel_buffers);
	hipFree((*cudaContext)->wafel_positions);
	hipFree((*cudaContext)->spec_buffer);

	hipFree(framebuffer_tex_out);

	for (unsigned int i = 0; i < (*cudaContext)->fringe_context->display_options.num_heads / 2; i++)
	{
		hipGraphicsUnregisterResource((*cudaContext)->fringe_cuda_resources[i]);
	}

	hipGraphicsUnregisterResource((*cudaContext)->stereogram_depth_cuda_resource);
	hipGraphicsUnregisterResource((*cudaContext)->stereogram_rgba_cuda_resource);

	if (*cudaContext != NULL)
	{
		if ((*cudaContext)->fringe_cuda_resources)
		{
			free((*cudaContext)->fringe_cuda_resources);
			(*cudaContext)->fringe_cuda_resources = NULL;
		}

		if ((*cudaContext)->gpu_properties)
		{
			free((*cudaContext)->gpu_properties);
			(*cudaContext)->gpu_properties = NULL;
		}

		free(*cudaContext);
		*cudaContext = NULL;
	}
};

void dscp4_fringe_cuda_ComputeFringe(dscp4_fringe_cuda_context_t* cudaContext)
{
	//void **output;
	//size_t * outputSizes;

	hipChannelFormatDesc rgbaTexDesc;
	rgbaTexDesc.x = 8;
	rgbaTexDesc.y = 8;
	rgbaTexDesc.z = 8;
	rgbaTexDesc.w = 8;
	rgbaTexDesc.f = hipChannelFormatKindUnsigned;

	hipArray_t viewsetRGBAArray;
	hipArray_t * framebufferArrays = NULL;
	void * viewsetDepthArray = NULL;
	size_t viewsetDepthArraySize;

	hipError_t error = hipSuccess;

	error = hipGraphicsMapResources(1, &cudaContext->stereogram_rgba_cuda_resource, 0);
	if(error != hipSuccess)
		printf("ERROR Mapping stereogram RGBA CUDA graphics resource\n");

	error = hipGraphicsMapResources(1, &cudaContext->stereogram_depth_cuda_resource, 0);
	if(error != hipSuccess)
		printf("ERROR Mapping stereogram DEPTH CUDA graphics resource\n");

	error = hipGraphicsSubResourceGetMappedArray(&viewsetRGBAArray, cudaContext->stereogram_rgba_cuda_resource, 0, 0);
	if (error != hipSuccess)
		printf("ERROR Mapping stereogram COLOR texture CUDA graphics resource\n");

	error = hipBindTextureToArray(&viewset_color_in, viewsetRGBAArray, &rgbaTexDesc);

	error = hipGraphicsResourceGetMappedPointer((void**)&viewsetDepthArray, &viewsetDepthArraySize, cudaContext->stereogram_depth_cuda_resource);
	if(error != hipSuccess)
		printf("ERROR Getting stereogram DEPTH CUDA graphics resource mapped pointer\n");


	//output = (void**)malloc(sizeof(void*)* cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers);
	//outputSizes = (size_t*)malloc(sizeof(size_t)* cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers);

	hipMalloc((void**)framebufferArrays, sizeof(hipArray_t)* cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers);

	for (unsigned int i = 0; i < cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers; i++)
	{
		//error = hipGraphicsMapResources(1, (hipGraphicsResource_t*)(&cudaContext->fringe_cuda_resources[i]), 0);
		//if(error != hipSuccess)
		//	printf("ERROR Mapping CUDA fringe texture buffer %i\n", i);


		//error = hipGraphicsResourceGetMappedPointer(&output[i], &outputSizes[i], cudaContext->fringe_cuda_resources[i]);
		//if (error != hipSuccess)
		//	printf("ERROR Getting fringe texture buffer %i CUDA mapped pointer\n", i);

		error = hipGraphicsSubResourceGetMappedArray(&framebufferArrays[i], cudaContext->fringe_cuda_resources[i],0,0);
		if (error != hipSuccess)
			printf("ERROR Getting framebuffer %i CUDA array\n", i);

		error = hipBindTextureToArray(&framebuffer_tex_out[i], framebufferArrays[i], &rgbaTexDesc);
		if (error != hipSuccess)
			printf("ERROR Binding framebuffer texture %i  to CUDA array\n", i);

	}

	//// run kernel here
	//for (unsigned int i = 0; i < cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers; i++)
	//{



		dim3 threadsPerBlock(
			cudaContext->fringe_context->algorithm_options.cuda_block_dimensions[0],
			cudaContext->fringe_context->algorithm_options.cuda_block_dimensions[1]
			);
		dim3 numBlocks(
			cudaContext->fringe_context->algorithm_options.cache.cuda_number_of_blocks[0],
			cudaContext->fringe_context->algorithm_options.cache.cuda_number_of_blocks[1]
			);

		computeFringe << <numBlocks, threadsPerBlock >> >(
			(unsigned char*)cudaContext->spec_buffer,
			(const float*)viewsetDepthArray,
			cudaContext->fringe_context->algorithm_options.num_wafels_per_scanline,
			cudaContext->fringe_context->algorithm_options.num_scanlines,
			cudaContext->fringe_context->algorithm_options.cache.stereogram_res_x,
			cudaContext->fringe_context->algorithm_options.cache.stereogram_res_y,
			cudaContext->fringe_context->algorithm_options.cache.stereogram_num_tiles_x,
			cudaContext->fringe_context->algorithm_options.cache.stereogram_num_tiles_y,
			cudaContext->fringe_context->algorithm_options.cache.fringe_buffer_res_x,
			cudaContext->fringe_context->algorithm_options.cache.fringe_buffer_res_y,
			cudaContext->wafel_buffers,
			cudaContext->wafel_positions,
			cudaContext->fringe_context->algorithm_options.cache.k_r,
			cudaContext->fringe_context->algorithm_options.cache.k_g,
			cudaContext->fringe_context->algorithm_options.cache.k_b,
			cudaContext->fringe_context->algorithm_options.cache.upconvert_const_r,
			cudaContext->fringe_context->algorithm_options.cache.upconvert_const_g,
			cudaContext->fringe_context->algorithm_options.cache.upconvert_const_b,
			cudaContext->fringe_context->algorithm_options.cache.num_samples_per_wafel,
			cudaContext->fringe_context->algorithm_options.cache.sample_pitch,
			cudaContext->fringe_context->algorithm_options.cache.z_span,
			cudaContext->fringe_context->algorithm_options.cache.z_offset,
			cudaContext->fringe_context->display_options.num_aom_channels,
			cudaContext->fringe_context->display_options.head_res_y_spec,
			cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers
			);
	//}g

	error = hipUnbindTexture(&viewset_color_in);

	//write texture outputs here

	for (unsigned int i = 0; i < cudaContext->fringe_context->display_options.num_heads; i++)
	{
		hipMemcpyToArray(framebufferArrays[i], 0, 0, cudaContext->spec_buffer, cudaContext->fringe_context->algorithm_options.cache.fringe_buffer_res_x * cudaContext->fringe_context->algorithm_options.cache.fringe_buffer_res_y *4, hipMemcpyDeviceToDevice);
	}

	for (unsigned int i = 0; i < cudaContext->fringe_context->algorithm_options.cache.num_fringe_buffers; i++)
	{
		hipUnbindTexture(framebuffer_tex_out[i]);
		error = hipGraphicsUnmapResources(1, (hipGraphicsResource_t*)(&cudaContext->fringe_cuda_resources[i]), 0);
		if(error != hipSuccess)
			printf("ERROR Unmapping CUDA fringe buffer %i resource\n", i);

	}

	error = hipGraphicsUnmapResources(1, &cudaContext->stereogram_rgba_cuda_resource, 0);
	if(error != hipSuccess)
		printf("ERROR Unmapping viewset RGBA CUDA graphics resource\n");

	error = hipGraphicsUnmapResources(1, &cudaContext->stereogram_depth_cuda_resource, 0);
	if(error != hipSuccess)
		printf("ERROR Unmapping viewset DEPTH CUDA graphics resource\n");


	//free(output);
	//free(outputSizes);
	free(framebufferArrays);
};

__global__ void computeFringe(
	unsigned char* framebuffer_out,
	const float* viewset_depth_in,
	const unsigned int num_wafels_per_scanline,
	const unsigned int num_scanlines,
	const unsigned int viewset_res_x,
	const unsigned int viewset_res_y,
	const unsigned int viewset_num_tiles_x,
	const unsigned int viewset_num_tiles_y,
	const unsigned int framebuffer_res_x,
	const unsigned int framebuffer_res_y,
	unsigned char* wafel_buffer,
	float* wafel_position,
	const float K_R,
	const float K_G,
	const float K_B,
	const float UPCONVERT_CONST_R,
	const float UPCONVERT_CONST_G,
	const float UPCONVERT_CONST_B,
	const unsigned int NUM_SAMPLES_PER_WAFEL,
	const float SAMPLE_PITCH,
	const float Z_SPAN,
	const float Z_OFFSET,
	const unsigned int NUM_AOM_CHANNELS,
	const unsigned int HEAD_RES_Y_SPEC,
	const unsigned int NUM_BUFFERS
	)
{

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < num_wafels_per_scanline && y < num_scanlines)
	{
		// offset of wafel samples/position buffer
		const unsigned int wafel_offset = x*y*NUM_SAMPLES_PER_WAFEL;
		const float num_views = (viewset_num_tiles_x * viewset_num_tiles_y);

		for (int i = 0; i < NUM_SAMPLES_PER_WAFEL; i++)
		{
			wafel_position[i + wafel_offset] = (-(float)ceil((double)(num_wafels_per_scanline / 2.f)) + i) * SAMPLE_PITCH + x;
		}

		for (unsigned int color_chan = 0; color_chan < 3; color_chan++)
		{
			x = (blockIdx.x * blockDim.x) + threadIdx.x;
			y = (blockIdx.y * blockDim.y) + threadIdx.y;

			float k = (color_chan == 0 ? K_R : color_chan == 1 ? K_G : K_B);
			float up_const = (color_chan == 0 ? UPCONVERT_CONST_R : color_chan == 1 ? UPCONVERT_CONST_G : UPCONVERT_CONST_B);

			for (unsigned int vy = 0; vy < viewset_num_tiles_y; vy++)
			{
				for (unsigned int vx = 0; vx < viewset_num_tiles_x; vx++)
				{
					float d = (viewset_depth_in[y * viewset_res_x + x] - 0.5) * Z_SPAN + Z_OFFSET;
					float4 color = tex2D(viewset_color_in, x, y);
					float c = 255.f*(color_chan == 0 ? color.x : color_chan == 1 ? color.y : color.z);

					for (int i = 0; i < NUM_SAMPLES_PER_WAFEL; i++)
					{
						wafel_buffer[wafel_offset + i] += c / num_views * cos(k * sqrt(pow((float)((int)wafel_position[wafel_offset + i] - (int)x), (float)2) + pow(d, (float)2)) - d + wafel_position[wafel_offset+i] * up_const);
					}
					//framebuffer_out[(y * framebuffer_res_x * 4) + (x * 4)] = viewset_depth_in[y * viewset_res_x + x] * 255.f;
					//framebuffer_out[(y * framebuffer_res_x * 4) + (x * 4 + 1)] = 0;
					//framebuffer_out[(y * framebuffer_res_x * 4) + (x * 4 + 2)] = 0;
					x += num_wafels_per_scanline;
				}
				x = (blockIdx.x * blockDim.x) + threadIdx.x;
				y += num_scanlines;
			}
		}

		x = (blockIdx.x * blockDim.x) + threadIdx.x;
		y = (blockIdx.y * blockDim.y) + threadIdx.y;

		int which_frame_buf = (y % NUM_AOM_CHANNELS);
		int which_hololine = y / NUM_AOM_CHANNELS;
//		int which_frameline = (float)x / (framebuffer_res_x / NUM_SAMPLES_PER_WAFEL);
//		int which_wafel = x - (which_frameline * (framebuffer_res_x / NUM_SAMPLES_PER_WAFEL));

		for (int i = 0; i < NUM_SAMPLES_PER_WAFEL; i++)
		{
			framebuffer_out[
				which_frame_buf / NUM_BUFFERS * framebuffer_res_x * HEAD_RES_Y_SPEC * 4
					+ which_hololine * (((NUM_SAMPLES_PER_WAFEL * num_wafels_per_scanline) / framebuffer_res_x) * framebuffer_res_x * 4)
					+ NUM_SAMPLES_PER_WAFEL * 4 * x
					+ which_frame_buf % 3 + 4 * i
			] = wafel_buffer[wafel_offset + i];
		}

	}
}
